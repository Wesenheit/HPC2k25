
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>

#define RADIUS        16
#define NUM_ELEMENTS  262144 

static void handleError(hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define cudaCheck( err ) (handleError(err, __FILE__, __LINE__ ))

__global__ void stencil_1d(int *in, int *out) {
    int i = threadIdx.x + blockIdx.x * blockDim.x; 

    if (i > RADIUS && i < NUM_ELEMENTS - RADIUS)
    {
        for (int idx = -RADIUS; idx <= RADIUS;idx++)
        {
            out[i] += in[i+idx];
        }
    }
}

void cpu_stencil_1d(int *in, int *out) {

    for (int i = RADIUS; i < NUM_ELEMENTS - RADIUS; i++)
    {
        for (int idx = -RADIUS; idx <= RADIUS;idx++)
        {
            out[i] += in[i+idx];
        }
        
    }
}

int main() {
    int *in,*out;
    int *in_gpu,*out_gpu;
    int *in_prim,*out_prim;
    in = new int[NUM_ELEMENTS];
    out = new int[NUM_ELEMENTS];
    in_prim = new int[NUM_ELEMENTS];
    out_prim = new int[NUM_ELEMENTS];
    
    cudaCheck(hipMalloc(&in_gpu,sizeof(int)*NUM_ELEMENTS));
    cudaCheck(hipMalloc(&out_gpu,sizeof(int)*NUM_ELEMENTS));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
   


    // time 1
    hipEventRecord( start, 0 );
    
    cudaCheck(hipMemcpy(in,in_gpu,sizeof(int)*NUM_ELEMENTS,hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(out,out_gpu,sizeof(int)*NUM_ELEMENTS,hipMemcpyDeviceToHost));
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop);
    printf("GPU transfer time (in):  %3.4f ms\n", elapsedTime);

    const int thread_num = 256;
    dim3 thread(thread_num);
    dim3 block(NUM_ELEMENTS/thread_num);
    
    //Warmup
    
    hipEventRecord( start, 0 );
    stencil_1d<<<block,thread>>>(in_gpu,out_gpu);
    cudaCheck(hipPeekAtLastError());
  
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime( &elapsedTime, start, stop);
    printf("GPU execution time:  %3.4f ms\n", elapsedTime);
    
    hipEventRecord( start, 0 );
    
    cudaCheck(hipMemcpy(in_gpu,in_prim,sizeof(int)*NUM_ELEMENTS,hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(out_gpu,out_prim,sizeof(int)*NUM_ELEMENTS,hipMemcpyHostToDevice));
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime( &elapsedTime, start, stop);
    printf("GPU transfer time (back):  %3.4f ms\n", elapsedTime);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);


    struct timespec cpu_start, cpu_stop;
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_start);

    cpu_stencil_1d(in, out);

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_stop);
    double result = (cpu_stop.tv_sec - cpu_start.tv_sec) * 1e3 + (cpu_stop.tv_nsec - cpu_start.tv_nsec) / 1e6;
    printf( "CPU execution time:  %3.4f ms\n", result);
    delete[] out;
    delete[] in;
    delete[] out_prim;
    delete[] in_prim;
    cudaCheck(hipFree(in_gpu));
    cudaCheck(hipFree(out_gpu));
    return 0;
}


